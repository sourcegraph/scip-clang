#include "hip/hip_runtime.h"
  // Initially based off kernel-call.cu in the Clang tests
//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^ definition [..] `<file>/kernelcall.cu`/
  // https://sourcegraph.com/github.com/llvm/llvm-project/-/blob/clang/test/SemaCUDA/kernel-call.cu
  
  #include "cuda_stub.h"
//         ^^^^^^^^^^^^^ reference [..] `<file>/cuda_stub.h`/
  
  __global__ void g1(int x) {}
//^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                ^^ definition [..] g1(d4f767463ce0a6b3).
//                       ^ definition local 0
  
  template <typename T> void t1(T arg) {
//                   ^ definition local 1
//                           ^^ definition [..] t1(9b289cee16747614).
//                              ^ reference local 1
//                                ^^^ definition local 2
    g1<<<arg, arg>>>(1);
//  ^^ reference [..] g1(d4f767463ce0a6b3).
//       ^^^ reference local 2
//            ^^^ reference local 2
  }
  
  void h1(int x) {}
//     ^^ definition [..] h1(d4f767463ce0a6b3).
//            ^ definition local 3
  int h2(int x) { return 1; }
//    ^^ definition [..] h2(7864480464b09eea).
//           ^ definition local 4
  
  int main(void) {
//    ^^^^ definition [..] main(b126dc7c1de90089).
    g1<<<1, 1>>>(42);
//  ^^ reference [..] g1(d4f767463ce0a6b3).
//       ^ reference [..] dim3#dim3(6df00707c193238d).
//          ^ reference [..] dim3#dim3(6df00707c193238d).
    g1(42); // expected-error {{call to global function 'g1' not configured}}
    g1<<<1>>>(42); // expected-error {{too few execution configuration arguments to kernel function call}}
    g1<<<1, 1, 0, 0, 0>>>(42); // expected-error {{too many execution configuration arguments to kernel function call}}
  
    t1(1);
//  ^^ reference [..] t1(9b289cee16747614).
  
    h1<<<1, 1>>>(42); // expected-error {{kernel call to non-global function 'h1'}}
//  ^^ reference [..] h1(d4f767463ce0a6b3).
  
    int (*fp)(int) = h2;
//        ^^ definition local 5
//                   ^^ reference [..] h2(7864480464b09eea).
    fp<<<1, 1>>>(42); // expected-error {{must have void return type}}
//  ^^ reference local 5
  
    g1<<<undeclared, 1>>>(42); // expected-error {{use of undeclared identifier 'undeclared'}}
//  ^^ reference [..] g1(d4f767463ce0a6b3).
  }
  
  // Make sure we can call static member kernels.
  template <typename > struct a0 {
//                            ^^ definition [..] a0#
    template <typename T> static __global__ void Call(T);
//                     ^ definition local 6
//                               ^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                                               ^^^^ reference [..] a0#Call(b07662a27bd562f9).
//                                                    ^ reference local 6
  };
  struct a1 {
//       ^^ definition [..] a1#
    template <typename T> static __global__ void Call(T);
//                     ^ definition local 7
//                               ^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                                               ^^^^ reference [..] a1#Call(9b289cee16747614).
//                                                    ^ reference local 7
  };
  template <typename T> struct a2 {
//                   ^ definition local 8
//                             ^^ definition [..] a2#
    static __global__ void Call(T);
//         ^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                         ^^^^ reference [..] a2#Call(9b289cee16747614).
//                              ^ reference local 8
  };
  struct a3 {
//       ^^ definition [..] a3#
    static __global__ void Call(int);
//         ^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                         ^^^^ reference [..] a3#Call(d4f767463ce0a6b3).
    static __global__ void Call(void*);
//         ^^^^^^^^^^ reference [..] `cuda_stub.h:12:9`!
//                         ^^^^ reference [..] a3#Call(5d22bdacc48458e8).
  };
  
  struct b {
//       ^ definition [..] b#
    template <typename c> void d0(c arg) {
//                     ^ definition local 9
//                             ^^ definition [..] b#d0(9b289cee16747614).
//                                ^ reference local 9
//                                  ^^^ definition local 10
      a0<c>::Call<<<0, 0>>>(arg);
//    ^^ reference [..] a0#
//                  ^ reference [..] dim3#dim3(6df00707c193238d).
//                     ^ reference [..] dim3#dim3(6df00707c193238d).
//                          ^^^ reference local 10
      a1::Call<<<0,0>>>(arg);
//    ^^ reference [..] a1#
//               ^ reference [..] dim3#dim3(6df00707c193238d).
//                 ^ reference [..] dim3#dim3(6df00707c193238d).
//                      ^^^ reference local 10
      a2<c>::Call<<<0,0>>>(arg);
//    ^^ reference [..] a2#
//                  ^ reference [..] dim3#dim3(6df00707c193238d).
//                    ^ reference [..] dim3#dim3(6df00707c193238d).
//                         ^^^ reference local 10
      a3::Call<<<0, 0>>>(arg);
//    ^^ reference [..] a3#
//               ^ reference [..] dim3#dim3(6df00707c193238d).
//                  ^ reference [..] dim3#dim3(6df00707c193238d).
//                       ^^^ reference local 10
    }
    void d1(void* arg) {
//       ^^ definition [..] b#d1(5d22bdacc48458e8).
//                ^^^ definition local 11
      a0<void*>::Call<<<0, 0>>>(arg);
//    ^^ reference [..] a0#
//               ^^^^ reference [..] a0#Call(9b289cee16747614).
//                      ^ reference [..] dim3#dim3(6df00707c193238d).
//                         ^ reference [..] dim3#dim3(6df00707c193238d).
//                              ^^^ reference local 11
      a1::Call<<<0,0>>>(arg);
//    ^^ reference [..] a1#
//        ^^^^ reference [..] a1#Call(9b289cee16747614).
//               ^ reference [..] dim3#dim3(6df00707c193238d).
//                 ^ reference [..] dim3#dim3(6df00707c193238d).
//                      ^^^ reference local 11
      a2<void*>::Call<<<0,0>>>(arg);
//    ^^ reference [..] a2#
//               ^^^^ reference [..] a2#Call(9b289cee16747614).
//                      ^ reference [..] dim3#dim3(6df00707c193238d).
//                        ^ reference [..] dim3#dim3(6df00707c193238d).
//                             ^^^ reference local 11
      a3::Call<<<0, 0>>>(arg);
//    ^^ reference [..] a3#
//        ^^^^ reference [..] a3#Call(5d22bdacc48458e8).
//               ^ reference [..] dim3#dim3(6df00707c193238d).
//                  ^ reference [..] dim3#dim3(6df00707c193238d).
//                       ^^^ reference local 11
    }
    void e() { d0(1); }
//       ^ definition [..] b#e(49f6e7a06ebc5aa8).
//             ^^ reference [..] b#d0(d4f767463ce0a6b3).
  };
